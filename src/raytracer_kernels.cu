
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <chrono>
#include <cmath>
#include <vector>
#include <algorithm>

#include <hiprand/hiprand_kernel.h>
#include <float.h>

using Int = unsigned long long;
const Int Atomic_reduce_const = (Int)(-1LL);

// using Int = unsigned int;
// const Int Atomic_reduce_const = (Int)(-1);

// using Float = double;
// const Float Float_epsilon = DBL_EPSILON;
// constexpr int block_size = 768;
// constexpr int grid_size = 64;

using Float = float;
const Float Float_epsilon = FLT_EPSILON;
constexpr int block_size = 512;
constexpr int grid_size = 64;

struct Vector
{
    Float x;
    Float y;
    Float z;

};

static inline __device__
Vector operator*(const Vector v, const Float s) { return Vector{s*v.x, s*v.y, s*v.z}; }
static inline __device__
Vector operator*(const Float s, const Vector v) { return Vector{s*v.x, s*v.y, s*v.z}; }
static inline __device__
Vector operator-(const Vector v1, const Vector v2) { return Vector{v1.x-v2.x, v1.y-v2.y, v1.z-v2.z}; }
static inline __device__
Vector operator+(const Vector v1, const Vector v2) { return Vector{v1.x+v2.x, v1.y+v2.y, v1.z+v2.z}; }


struct Optics_ext
{
    Float gas;
    Float cloud;
};

struct Optics_scat
{
    Float ssa;
    Float asy;
};



__device__
Vector cross(const Vector v1, const Vector v2)
{
    return Vector{
            v1.y*v2.z - v1.z*v2.y,
            v1.z*v2.x - v1.x*v2.z,
            v1.x*v2.y - v1.y*v2.x};
}


__device__
Float dot(const Vector v1, const Vector v2)
{
    return v1.x*v2.x + v1.y*v2.y + v1.z*v1.z;
}

__device__
Float norm(const Vector v) { return sqrt(v.x*v.x + v.y*v.y + v.z*v.z); }


__device__
Vector normalize(const Vector v)
{
    const Float length = norm(v);
    return Vector{ v.x/length, v.y/length, v.z/length};
}

enum class Photon_kind { Direct, Diffuse };

struct Photon
{
    Vector position;
    Vector direction;
    Photon_kind kind;
};


__device__
Float pow2(const Float d) { return d*d; }


#define cuda_safe_call(ans) { gpu_assert((ans), __FILE__, __LINE__); }

inline void gpu_assert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


template<typename T>
T* allocate_gpu(const int length)
{
    T* data_ptr = nullptr;
    cuda_safe_call(hipMalloc((void **) &data_ptr, length*sizeof(T)));

    return data_ptr;
}


template<typename T>
void free_gpu(T*& data_ptr)
{
    cuda_safe_call(hipFree(data_ptr));
    data_ptr = nullptr;
}


template<typename T>
void copy_to_gpu(T* gpu_data, const T* cpu_data, const int length)
{
    cuda_safe_call(hipMemcpy(gpu_data, cpu_data, length*sizeof(T), hipMemcpyHostToDevice));
}


template<typename T>
void copy_from_gpu(T* cpu_data, const T* gpu_data, const int length)
{
    cuda_safe_call(hipMemcpy(cpu_data, gpu_data, length*sizeof(T), hipMemcpyDeviceToHost));
}


__device__
Float rayleigh(const Float random_number)
{
    const Float q = Float(4.)*random_number - Float(2.);
    const Float d = Float(1.) + pow2(q);
    const Float u = pow(-q + sqrt(d), Float(1./3.));
    return u - Float(1.)/u;
}


__device__
Float henyey(const Float g, const Float random_number)
{
    const Float a = pow2(Float(1.) - pow2(g));
    const Float b = Float(2.)*g*pow2(Float(2.)*random_number*g + Float(1.) - g);
    const Float c = -g/Float(2.) - Float(1.)/(Float(2.)*g);
    return Float(-1.)*(a/b) - c;
}


__device__
Float sample_tau(const Float random_number)
{
    // Prevent log(0) possibility.
    return Float(-1.)*log(-random_number + Float(1.) + Float_epsilon);
}


__device__
inline int float_to_int(const float s_size, const float ds, const int ntot_max)
{
    const int ntot = static_cast<int>(s_size / ds);
    return ntot < ntot_max ? ntot : ntot_max-1;
}

__device__
inline void reset_photon(
        Photon& photon, Int& photons_shot, Int* __restrict__ const toa_down_count,
        const unsigned int random_number_x, const unsigned int random_number_y,
        const Float x_size, const Float y_size, const Float z_size,
        const Float dx_grid, const Float dy_grid, const Float dz_grid,
        const Float dir_x, const Float dir_y, const float dir_z,
        const bool generation_completed,
        const int itot, const int jtot)
{
    if (!generation_completed)
    {
        const int i = random_number_x / static_cast<unsigned int>((1ULL << 32) / itot);
        const int j = random_number_y / static_cast<unsigned int>((1ULL << 32) / jtot);

        photon.position.x = x_size * random_number_x / (1ULL << 32);
        photon.position.y = y_size * random_number_y / (1ULL << 32);
        photon.position.z = z_size;

        photon.direction.x = dir_x;
        photon.direction.y = dir_y;
        photon.direction.z = dir_z;

        photon.kind = Photon_kind::Direct;

        const int ij = i + j*itot;
        atomicAdd(&toa_down_count[ij], 1);
    }
}


template<typename T>
struct Random_number_generator
{
    __device__ Random_number_generator(unsigned int tid)
    {
        hiprand_init(tid, tid, 0, &state);
    }

    __device__ T operator()();

    hiprandState state;
};


template<>
__device__ double Random_number_generator<double>::operator()()
{
    return 1. - hiprand_uniform_double(&state);
}


template<>
__device__ float Random_number_generator<float>::operator()()
{
    return 1.f - hiprand_uniform(&state);
}


struct Quasi_random_number_generator_2d
{
    __device__ Quasi_random_number_generator_2d(
            hiprandDirectionVectors32_t* vectors, unsigned int* constants, unsigned int offset)
    {
        hiprand_init(vectors[0], constants[0], offset, &state_x);
        hiprand_init(vectors[1], constants[1], offset, &state_y);
    }

    __device__ unsigned int x() { return hiprand(&state_x); }
    __device__ unsigned int y() { return hiprand(&state_y); }

    hiprandStateScrambledSobol32_t state_x;
    hiprandStateScrambledSobol32_t state_y;
};


__device__
inline void write_photon_out(Int* field_out, Int& photons_shot, const Int inc)
{
    photons_shot += inc;
    atomicAdd(field_out, 1);
}

__global__
void cloud_mask_kernel(
    const Optics_scat* __restrict__ ssa_asy,
    Int* __restrict__ cloud_mask_v,
    Float* __restrict__ cloud_dims,
    const Float dz_grid,
    const int itot, const int jtot, const int ktot)
{
    const int k = blockDim.x * blockIdx.x + threadIdx.x;
    if (k < ktot)
    {
        cloud_mask_v[k] = 0;
        for (int j=0; j<jtot; ++j)
            for (int i=0; i<jtot; ++i)
            {
                const int ijk = i + j*itot + k*jtot*itot;
                if (ssa_asy[ijk].asy > 0)
                {
                    cloud_mask_v[k] = 1;
                    return;
                }
            }
    }
    __syncthreads();
    if (k==0)
    {
        for (int i=0; i<ktot; ++i)
            if (cloud_mask_v[i]==1)
            {
                cloud_dims[0] = i*dz_grid;
                return;
            }
    }
    if (k==1)
    {
        for (int i=ktot; i>0; --i)
            if (cloud_mask_v[i]==1)
            {
                cloud_dims[1] = (i+1)*dz_grid;
                return;
            }
    }
}

__global__
void ray_tracer_kernel(
        const Int photons_to_shoot,
        Photon* __restrict__ photons,
        Int* __restrict__ toa_down_count,
        Int* __restrict__ toa_up_count,
        Int* __restrict__ surface_down_direct_count,
        Int* __restrict__ surface_down_diffuse_count,
        Int* __restrict__ surface_up_count,
        Int* __restrict__ atmos_direct_count,
        Int* __restrict__ atmos_diffuse_count,
        const Optics_ext* __restrict__ k_ext, const Optics_scat* __restrict__ ssa_asy,
        const Float k_ext_null_cld, const Float k_ext_null_gas,
        const Float surface_albedo,
        const Float x_size, const Float y_size, const Float z_size,
        const Float dx_grid, const Float dy_grid, const Float dz_grid,
        const Float dir_x, const Float dir_y, const Float dir_z,
        const int itot, const int jtot, const int ktot,
        hiprandDirectionVectors32_t* qrng_vectors, unsigned int* qrng_constants,
        const Float* __restrict__ cloud_dims)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;

    Random_number_generator<Float> rng(n);
    Quasi_random_number_generator_2d qrng(qrng_vectors, qrng_constants, n * photons_to_shoot);

    const Float cloud_min = cloud_dims[0];
    const Float cloud_max = cloud_dims[1];
    const Float s_min = x_size * Float_epsilon;

    // Set up the initial photons.
    const bool completed = false;
    Int photons_shot = 0;

    reset_photon(
            photons[n], photons_shot, toa_down_count,
            qrng.x(), qrng.y(),
            x_size, y_size, z_size,
            dx_grid, dy_grid, dz_grid,
            dir_x, dir_y, dir_z,
            completed,
            itot, jtot);

    Float tau;
    bool surface_exit = false;
    bool toa_exit = false;
    bool transition = false;


    while (photons_shot < photons_to_shoot)
    {

        const bool photon_generation_completed = (photons_shot == photons_to_shoot - 1);
        const bool photon_in_cloud = (photons[n].position.z >= cloud_min && photons[n].position.z <= cloud_max);

        const Float k_ext_null = photon_in_cloud ? k_ext_null_cld : k_ext_null_gas;

        if (!transition)
            tau = sample_tau(rng());

        const Float dn = max(Float_epsilon, sample_tau(rng()) / k_ext_null);
        Float dx = photons[n].direction.x * dn;
        Float dy = photons[n].direction.y * dn;
        Float dz = photons[n].direction.z * dn;

        surface_exit = false;
        toa_exit = false;
        transition = false;

        if (photon_in_cloud)
        {
            const double fac = (photons[n].direction.z > 0) ? (cloud_max-photons[n].position.z) / dz : (cloud_min - photons[n].position.z) / dz;

            if (fac < 1)
            {
                dx *= fac;
                dy *= fac;
                dz *= fac;

                transition = true;

                if ( (photons[n].position.z == cloud_min) && (photons[n].direction.z < 0) )
                    photons[n].position.z -= s_min;

                if ( (photons[n].position.z == cloud_max) && (photons[n].direction.z > 0) )
                    photons[n].position.z += s_min;
            }
        }
        // photon above cloud layer, but about to cross it!
        else if ( (photons[n].position.z > cloud_max) && (photons[n].position.z + dz <= cloud_max) )
        {
            const Float fac = std::abs((photons[n].position.z - cloud_max) / dz);
            dx *= fac;
            dy *= fac;
            dz *= fac;

            transition = true;
        }

        // photon below cloud layer, but about to cross it! (if "constant_gas" is enabled)
        else if (photons[n].position.z < cloud_min && photons[n].position.z + dz >= cloud_min)
        {
            const Float fac = std::abs((photons[n].position.z - cloud_min) / dz);
            dx *= fac;
            dy *= fac;
            dz *= fac;

            transition = true;
        }

        if ((photons[n].position.z + dz) <= Float(0.))
        {
            const Float fac = abs(photons[n].position.z / dz);
            dx *= fac;
            dy *= fac;
            dz *= fac;

            surface_exit = true;
        }
        else if ((photons[n].position.z + dz) >= z_size)
        {
            const Float fac = abs((z_size - photons[n].position.z) / dz);
            dx *= fac;
            dy *= fac;
            dz *= fac;

            toa_exit = true;
        }

        photons[n].position.x += dx;
        photons[n].position.y += dy;
        photons[n].position.z += dz;

        // Cyclic boundary condition in x.
        photons[n].position.x = fmod(photons[n].position.x, x_size);
        if (photons[n].position.x < Float(0.))
            photons[n].position.x += x_size;

        // Cyclic boundary condition in y.
        photons[n].position.y = fmod(photons[n].position.y, y_size);
        if (photons[n].position.y < Float(0.))
            photons[n].position.y += y_size;

        // Handle the surface and top exits.
        const int i = float_to_int(photons[n].position.x, dx_grid, itot);
        const int j = float_to_int(photons[n].position.y, dy_grid, jtot);
        const int ij = i + j*itot;

        if (surface_exit)
        {
            if (photons[n].kind == Photon_kind::Direct)
                write_photon_out(&surface_down_direct_count[ij], photons_shot, 1);
            else if (photons[n].kind == Photon_kind::Diffuse)
                write_photon_out(&surface_down_diffuse_count[ij], photons_shot, 1);

            // Surface scatter if smaller than albedo, otherwise absorb
            if (rng() <= surface_albedo)
            {
                write_photon_out(&surface_up_count[ij], photons_shot, Atomic_reduce_const);

                const Float mu_surface = sqrt(rng());
                const Float azimuth_surface = Float(2.*M_PI)*rng();

                photons[n].direction.x = mu_surface*sin(azimuth_surface);
                photons[n].direction.y = mu_surface*cos(azimuth_surface);
                photons[n].direction.z = sqrt(Float(1.) - mu_surface*mu_surface + Float_epsilon);
                photons[n].kind = Photon_kind::Diffuse;
            }
            else
            {
                reset_photon(
                        photons[n], photons_shot, toa_down_count,
                        qrng.x(), qrng.y(),
                        x_size, y_size, z_size,
                        dx_grid, dy_grid, dz_grid,
                        dir_x, dir_y, dir_z,
                        photon_generation_completed,
                        itot, jtot);
            }
        }
        else if (toa_exit)
        {
            write_photon_out(&toa_up_count[ij], photons_shot, 1);

            reset_photon(
                    photons[n], photons_shot, toa_down_count,
                    qrng.x(), qrng.y(),
                    x_size, y_size, z_size,
                    dx_grid, dy_grid, dz_grid,
                    dir_x, dir_y, dir_z,
                    photon_generation_completed,
                    itot, jtot);
        }
        else if (transition)
        {
            tau -= dn * k_ext_null;
        }
        else
        {
            // Calculate the 3D index.
            const int k = float_to_int(photons[n].position.z, dz_grid, ktot);
            const int ijk = i + j*itot + k*itot*jtot;

            // Handle the action.
            const Float random_number = rng();

            // Null collision.
            if (random_number >= ((k_ext[ijk].gas + k_ext[ijk].cloud) / k_ext_null))
            {
            }
            // Scattering.
            else if (random_number <= ssa_asy[ijk].ssa * (k_ext[ijk].gas + k_ext[ijk].cloud) / k_ext_null)
            {
                const bool cloud_scatter = rng() < k_ext[ijk].cloud / (k_ext[ijk].gas + k_ext[ijk].cloud);

                const Float cos_scat = cloud_scatter ? henyey(ssa_asy[ijk].asy, rng()) : rayleigh(rng());
                const Float sin_scat = sqrt(Float(1.) - cos_scat*cos_scat + Float_epsilon);

                Vector t1{Float(0.), Float(0.), Float(0.)};
                if (fabs(photons[n].direction.x) < fabs(photons[n].direction.y))
                {
                    if (fabs(photons[n].direction.x) < fabs(photons[n].direction.z))
                        t1.x = Float(1.);
                    else
                        t1.z = Float(1.);
                }
                else
                {
                    if (fabs(photons[n].direction.y) < fabs(photons[n].direction.z))
                        t1.y = Float(1.);
                    else
                        t1.z = Float(1.);
                }
                t1 = normalize(t1 - photons[n].direction*dot(t1, photons[n].direction));
                Vector t2 = cross(photons[n].direction, t1);

                const Float phi = Float(2.*M_PI)*rng();

                photons[n].direction = cos_scat*photons[n].direction
                        + sin_scat*(sin(phi)*t1 + cos(phi)*t2);

                photons[n].kind = Photon_kind::Diffuse;
            }
            // Absorption.
            else
            {
                if (photons[n].kind == Photon_kind::Direct)
                    write_photon_out(&atmos_direct_count[ijk], photons_shot, 1);
                else
                    write_photon_out(&atmos_diffuse_count[ijk], photons_shot, 1);

                reset_photon(
                        photons[n], photons_shot, toa_down_count,
                        qrng.x(), qrng.y(),
                        x_size, y_size, z_size,
                        dx_grid, dy_grid, dz_grid,
                        dir_x, dir_y, dir_z,
                        photon_generation_completed,
                        itot, jtot);
            }
        }
    }
}
