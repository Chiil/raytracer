
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <chrono>
#include <cmath>
#include <vector>
#include <algorithm>

#include <hiprand/hiprand_kernel.h>
#include <float.h>

using Int = unsigned long long;
const Int Atomic_reduce_const = (Int)(-1LL);

// using Int = unsigned int;
// const Int Atomic_reduce_const = (Int)(-1);

// using Float = double;
// const Float Float_epsilon = DBL_EPSILON;
// constexpr int block_size = 768;
// constexpr int grid_size = 64;

using Float = float;
const Float Float_epsilon = FLT_EPSILON;
constexpr int block_size = 512;
constexpr int grid_size = 64;

constexpr Float w_thres = 0.5;

struct Vector
{
    Float x;
    Float y;
    Float z;

};

static inline __device__
Vector operator*(const Vector v, const Float s) { return Vector{s*v.x, s*v.y, s*v.z}; }
static inline __device__
Vector operator*(const Float s, const Vector v) { return Vector{s*v.x, s*v.y, s*v.z}; }
static inline __device__
Vector operator-(const Vector v1, const Vector v2) { return Vector{v1.x-v2.x, v1.y-v2.y, v1.z-v2.z}; }
static inline __device__
Vector operator+(const Vector v1, const Vector v2) { return Vector{v1.x+v2.x, v1.y+v2.y, v1.z+v2.z}; }


struct Optics_ext
{
    Float gas;
    Float cloud;
};

struct Optics_scat
{
    Float ssa;
    Float asy;
};



__device__
Vector cross(const Vector v1, const Vector v2)
{
    return Vector{
            v1.y*v2.z - v1.z*v2.y,
            v1.z*v2.x - v1.x*v2.z,
            v1.x*v2.y - v1.y*v2.x};
}


__device__
Float dot(const Vector v1, const Vector v2)
{
    return v1.x*v2.x + v1.y*v2.y + v1.z*v1.z;
}

__device__
Float norm(const Vector v) { return sqrt(v.x*v.x + v.y*v.y + v.z*v.z); }


__device__
Vector normalize(const Vector v)
{
    const Float length = norm(v);
    return Vector{ v.x/length, v.y/length, v.z/length};
}

enum class Photon_kind { Direct, Diffuse };

struct Photon
{
    Vector position;
    Vector direction;
    Photon_kind kind;
};


__device__
Float pow2(const Float d) { return d*d; }


#define cuda_safe_call(ans) { gpu_assert((ans), __FILE__, __LINE__); }

inline void gpu_assert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


template<typename T>
T* allocate_gpu(const int length)
{
    T* data_ptr = nullptr;
    cuda_safe_call(hipMalloc((void **) &data_ptr, length*sizeof(T)));

    return data_ptr;
}


template<typename T>
void free_gpu(T*& data_ptr)
{
    cuda_safe_call(hipFree(data_ptr));
    data_ptr = nullptr;
}


template<typename T>
void copy_to_gpu(T* gpu_data, const T* cpu_data, const int length)
{
    cuda_safe_call(hipMemcpy(gpu_data, cpu_data, length*sizeof(T), hipMemcpyHostToDevice));
}


template<typename T>
void copy_from_gpu(T* cpu_data, const T* gpu_data, const int length)
{
    cuda_safe_call(hipMemcpy(cpu_data, gpu_data, length*sizeof(T), hipMemcpyDeviceToHost));
}


__device__
Float rayleigh(const Float random_number)
{
    const Float q = Float(4.)*random_number - Float(2.);
    const Float d = Float(1.) + pow2(q);
    const Float u = pow(-q + sqrt(d), Float(1./3.));
    return u - Float(1.)/u;
}


__device__
Float henyey(const Float g, const Float random_number)
{
    const Float a = pow2(Float(1.) - pow2(g));
    const Float b = Float(2.)*g*pow2(Float(2.)*random_number*g + Float(1.) - g);
    const Float c = -g/Float(2.) - Float(1.)/(Float(2.)*g);
    return Float(-1.)*(a/b) - c;
}


__device__
Float sample_tau(const Float random_number)
{
    // Prevent log(0) possibility.
    return Float(-1.)*log(-random_number + Float(1.) + Float_epsilon);
}


__device__
inline int float_to_int(const float s_size, const float ds, const int ntot_max)
{
    const int ntot = static_cast<int>(s_size / ds);
    return ntot < ntot_max ? ntot : ntot_max-1;
}

__device__
inline void reset_photon(
        Photon& photon, Int& photons_shot, Float* __restrict__ const toa_down_count,
        const unsigned int random_number_x, const unsigned int random_number_y,
        const Float x_size, const Float y_size, const Float z_size,
        const Float dx_grid, const Float dy_grid, const Float dz_grid,
        const Float dir_x, const Float dir_y, const float dir_z,
        const bool generation_completed, Float& weight,
        const int itot, const int jtot)
{
    ++photons_shot;
    if (!generation_completed)
    {
        const int i = random_number_x / static_cast<unsigned int>((1ULL << 32) / itot);
        const int j = random_number_y / static_cast<unsigned int>((1ULL << 32) / jtot);

        photon.position.x = x_size * random_number_x / (1ULL << 32);
        photon.position.y = y_size * random_number_y / (1ULL << 32);
        photon.position.z = z_size;

        photon.direction.x = dir_x;
        photon.direction.y = dir_y;
        photon.direction.z = dir_z;

        photon.kind = Photon_kind::Direct;
        
        const int ij = i + j*itot;
        atomicAdd(&toa_down_count[ij], 1);
    
        weight = 1;

    }
}


template<typename T>
struct Random_number_generator
{
    __device__ Random_number_generator(unsigned int tid)
    {
        hiprand_init(tid, tid, 0, &state);
    }

    __device__ T operator()();

    hiprandState state;
};


template<>
__device__ double Random_number_generator<double>::operator()()
{
    return 1. - hiprand_uniform_double(&state);
}


template<>
__device__ float Random_number_generator<float>::operator()()
{
    return 1.f - hiprand_uniform(&state);
}


struct Quasi_random_number_generator_2d
{
    __device__ Quasi_random_number_generator_2d(
            hiprandDirectionVectors32_t* vectors, unsigned int* constants, unsigned int offset)
    {
        hiprand_init(vectors[0], constants[0], offset, &state_x);
        hiprand_init(vectors[1], constants[1], offset, &state_y);
    }

    __device__ unsigned int x() { return hiprand(&state_x); }
    __device__ unsigned int y() { return hiprand(&state_y); }

    hiprandStateScrambledSobol32_t state_x;
    hiprandStateScrambledSobol32_t state_y;
};


__device__
inline void write_photon_out(Float* field_out, const Float w)
{
    atomicAdd(field_out, w);
}

__global__
void cloud_mask_kernel(
    const Optics_scat* __restrict__ ssa_asy,
    Int* __restrict__ cloud_mask_v,
    Float* __restrict__ cloud_dims,
    const Float dz_grid,
    const int itot, const int jtot, const int ktot)
{
    const int k = blockDim.x * blockIdx.x + threadIdx.x;
    if (k < ktot)
    {
        cloud_mask_v[k] = 0;
        for (int j=0; j<jtot; ++j)
            for (int i=0; i<jtot; ++i)
            {
                const int ijk = i + j*itot + k*jtot*itot;
                if (ssa_asy[ijk].asy > 0)
                {
                    cloud_mask_v[k] = 1;
                    return;
                }
            }
    }
    __syncthreads();
    if (k==0)
    {
        for (int i=0; i<ktot; ++i)
            if (cloud_mask_v[i]==1)
            {
                cloud_dims[0] = i*dz_grid;
                return;
            }
    }
    if (k==1)
    {
        for (int i=ktot; i>0; --i)
            if (cloud_mask_v[i]==1)
            {
                cloud_dims[1] = (i+1)*dz_grid;
                return;
            }
    }
}

__global__
void ray_tracer_kernel(
        const Int photons_to_shoot,
        Float* __restrict__ toa_down_count,
        Float* __restrict__ toa_up_count,
        Float* __restrict__ surface_down_direct_count,
        Float* __restrict__ surface_down_diffuse_count,
        Float* __restrict__ surface_up_count,
        Float* __restrict__ atmos_direct_count,
        Float* __restrict__ atmos_diffuse_count,
        const Optics_ext* __restrict__ k_ext, const Optics_scat* __restrict__ ssa_asy,
        const Float k_ext_null_cld, const Float k_ext_null_gas,
        const Float surface_albedo,
        const Float x_size, const Float y_size, const Float z_size,
        const Float dx_grid, const Float dy_grid, const Float dz_grid,
        const Float dir_x, const Float dir_y, const Float dir_z, 
        const int itot, const int jtot, const int ktot,
        hiprandDirectionVectors32_t* qrng_vectors, unsigned int* qrng_constants,
        const Float* __restrict__ cloud_dims)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    Photon photon;
    Random_number_generator<Float> rng(n);
    Quasi_random_number_generator_2d qrng(qrng_vectors, qrng_constants, n * photons_to_shoot);

    const Float cloud_min = cloud_dims[0];
    const Float cloud_max = cloud_dims[1];
    const Float s_min = x_size * Float_epsilon;

    // Set up the initial photons.
    const bool completed = false;
    Int photons_shot = Atomic_reduce_const;
    Float weight;

    reset_photon(
            photon, photons_shot, toa_down_count,
            qrng.x(), qrng.y(),
            x_size, y_size, z_size,
            dx_grid, dy_grid, dz_grid,
            dir_x, dir_y, dir_z,
            completed, weight,
            itot, jtot);
    
    Float tau;
    bool surface_exit = false;
    bool toa_exit = false;
    bool transition = false;
    

    while (photons_shot < photons_to_shoot)
    {       
        const bool photon_generation_completed = (photons_shot == photons_to_shoot - 1);
        const bool photon_in_cloud = (photon.position.z >= cloud_min && photon.position.z <= cloud_max);

        const Float k_ext_null = photon_in_cloud ? k_ext_null_cld : k_ext_null_gas;
        if (!transition) tau = sample_tau(rng());

        const Float dn = max(Float_epsilon, sample_tau(rng()) / k_ext_null);
        Float dx = photon.direction.x * dn;
        Float dy = photon.direction.y * dn;
        Float dz = photon.direction.z * dn;
        
        surface_exit = false;
        toa_exit = false;
        transition = false;

        if (photon_in_cloud)
        {
            const Float fac = (photon.direction.z > 0 ? (cloud_max-photon.position.z)/dz : (cloud_min-photon.position.z)/dz);
            if (fac < 1)
            {
                dx *= fac;
                dy *= fac;
                dz *= fac;
                transition=true;

                if (((photon.position.z - cloud_min) < Float_epsilon) && (photon.direction.z < 0))
                        photon.position.z -= s_min;
                
                if (((cloud_max - photon.position.z) < Float_epsilon) && (photon.direction.z > 0))
                        photon.position.z += s_min;
            }
        }
        // photon above cloud layer, but about to cross it! 
        else if (photon.position.z > cloud_max && photon.position.z + dz <= cloud_max)
        {
            const Float fac = std::abs((photon.position.z - cloud_max) / dz);
            dx *= fac;
            dy *= fac;
            dz *= fac;
            transition=true;
        }
        
        // photon below cloud layer, but about to cross it! (if "constant_gas" is enabled)
        else if (photon.position.z < cloud_min && photon.position.z + dz >= cloud_min)
        {
            const Float fac = std::abs((photon.position.z - cloud_min) / dz);
            dx *= fac;
            dy *= fac;
            dz *= fac;
            transition=true;
        }

        if ((photon.position.z + dz) <= Float(0.))
        {
            const Float fac = abs(photon.position.z / dz);
            dx *= fac;
            dy *= fac;
            dz *= fac;

            surface_exit = true;
        }
        else if ((photon.position.z + dz) >= z_size)
        {
            const Float fac = abs((z_size - photon.position.z) / dz);
            dx *= fac;
            dy *= fac;
            dz *= fac;

            toa_exit = true;
        }

        photon.position.x += dx;
        photon.position.y += dy;
        photon.position.z += dz;

        // Cyclic boundary condition in x.
        photon.position.x = fmod(photon.position.x, x_size);
        if (photon.position.x < Float(0.))
            photon.position.x += x_size;

        // Cyclic boundary condition in y.
        photon.position.y = fmod(photon.position.y, y_size);
        if (photon.position.y < Float(0.))
            photon.position.y += y_size;

        // Handle the surface and top exits.
        const int i = float_to_int(photon.position.x, dx_grid, itot);
        const int j = float_to_int(photon.position.y, dy_grid, jtot);
        const int ij = i + j*itot;

        if (surface_exit)
        {
            // Add surface irradiance
            if (photon.kind == Photon_kind::Direct)
                write_photon_out(&surface_down_direct_count[ij], weight);
            else if (photon.kind == Photon_kind::Diffuse)
                write_photon_out(&surface_down_diffuse_count[ij], weight);

            // Update weights and add upward surface flux
            weight *= surface_albedo;
            write_photon_out(&surface_up_count[ij], weight);

            if (weight < w_thres)
                weight = (rng() > weight) ? Float(0.) : Float(1.);

            // only with nonzero weight continue ray tracing, else start new ray
            if (weight > Float(0.))
            {
                const Float mu_surface = sqrt(rng());
                const Float azimuth_surface = Float(2.*M_PI)*rng();

                photon.direction.x = mu_surface*sin(azimuth_surface);
                photon.direction.y = mu_surface*cos(azimuth_surface);
                photon.direction.z = sqrt(Float(1.) - mu_surface*mu_surface + Float_epsilon);
                photon.kind = Photon_kind::Diffuse;
            }
            else
            {
                reset_photon(
                        photon, photons_shot, toa_down_count,
                        qrng.x(), qrng.y(),
                        x_size, y_size, z_size,
                        dx_grid, dy_grid, dz_grid,
                        dir_x, dir_y, dir_z,
                        photon_generation_completed, weight,
                        itot, jtot);
            }
        }
        else if (toa_exit)
        {
            write_photon_out(&toa_up_count[ij], weight);
            reset_photon(
                    photon, photons_shot, toa_down_count,
                    qrng.x(), qrng.y(),
                    x_size, y_size, z_size,
                    dx_grid, dy_grid, dz_grid,
                    dir_x, dir_y, dir_z,
                    photon_generation_completed, weight,
                    itot, jtot);
        }
        else if (transition)
        {
            tau -= dn * k_ext_null;
        }
        else
        {
            // Calculate the 3D index.
            const int k = float_to_int(photon.position.z, dz_grid, ktot);
            const int ijk = i + j*itot + k*itot*jtot;

            // Handle the action.
            const Float random_number = rng();
            const Float k_ext_tot = k_ext[ijk].gas + k_ext[ijk].cloud;
            
            // Compute probability not being absorbed and store weighted absorption probability
            const Float f_no_abs = Float(1.) - (Float(1.) - ssa_asy[ijk].ssa) * (k_ext_tot/k_ext_null);
            if (photon.kind == Photon_kind::Direct)
                write_photon_out(&atmos_direct_count[ijk], weight*(1-f_no_abs));
            else
                write_photon_out(&atmos_diffuse_count[ijk], weight*(1-f_no_abs));
            

            // Update weights (see Iwabuchi 2006: https://doi.org/10.1175/JAS3755.1)
            weight *= f_no_abs;
            if (weight < w_thres)
                weight = (rng() > weight) ? Float(0.) : Float(1.);

            // only with nonzero weight continue ray tracing, else start new ray
            if (weight > Float(0.))
            {
                // Null collision.
                if (random_number >= ssa_asy[ijk].ssa / (ssa_asy[ijk].ssa - Float(1.) + k_ext_null / k_ext_tot))
                {
                }
                // Scattering.
                else
                {
                    const bool cloud_scatter = rng() < (k_ext[ijk].cloud / k_ext_tot);
                    const Float cos_scat = cloud_scatter ? henyey(ssa_asy[ijk].asy, rng()) : rayleigh(rng());
                    const Float sin_scat = sqrt(Float(1.) - cos_scat*cos_scat + Float_epsilon);

                    Vector t1{Float(0.), Float(0.), Float(0.)};
                    if (fabs(photon.direction.x) < fabs(photon.direction.y))
                    {
                        if (fabs(photon.direction.x) < fabs(photon.direction.z))
                            t1.x = Float(1.);
                        else
                            t1.z = Float(1.);
                    }
                    else
                    {
                        if (fabs(photon.direction.y) < fabs(photon.direction.z))
                            t1.y = Float(1.);
                        else
                            t1.z = Float(1.);
                    }
                    t1 = normalize(t1 - photon.direction*dot(t1, photon.direction));
                    Vector t2 = cross(photon.direction, t1);

                    const Float phi = Float(2.*M_PI)*rng();

                    photon.direction = cos_scat*photon.direction
                            + sin_scat*(sin(phi)*t1 + cos(phi)*t2);

                    photon.kind = Photon_kind::Diffuse;
                }
            }
            else
            {
                reset_photon(
                        photon, photons_shot, toa_down_count,
                        qrng.x(), qrng.y(),
                        x_size, y_size, z_size,
                        dx_grid, dy_grid, dz_grid,
                        dir_x, dir_y, dir_z,
                        photon_generation_completed, weight,
                        itot, jtot);
    
            }
        }
    }
}


